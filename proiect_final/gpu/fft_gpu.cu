#include "hip/hip_runtime.h"
#include <stdio.h>
#include <inttypes.h>
#include <hip/hip_cooperative_groups.h>

#include "../shared_defs.h"

using namespace cooperative_groups;

__device__ uint32_t kernel_log_of_pow_of_2(uint32_t num) {
    uint32_t l;
    for (l = 0; l < 32; l++) {
        if (num&1) {
            return l;
        }
        num >>= 1;
    }
}

__global__ void complex_fft_iteration(float *real, float *imag, const uint32_t N,
                            const uint32_t index_multiplier) {
    int iterations = kernel_log_of_pow_of_2(N);
    float a_real, a_imag;
    uint32_t u, pow_2_i;
    float sin_u, cos_u;
    grid_group g = this_grid();
    int n = g.thread_index().x;

    for (int i = 1; i <= iterations; i++) {
        pow_2_i = 1 << i;
        u = n / (N/pow_2_i);
        sin_u = sin(-2 * (u * pi / pow_2_i));
        cos_u = cos(-2 * (u * pi / pow_2_i));

        a_real =
            real[(n + u*N/pow_2_i) % N * index_multiplier] +
            real[(n + u*N/pow_2_i + N/pow_2_i) % N * index_multiplier]*cos_u -
            imag[(n + u*N/pow_2_i + N/pow_2_i) % N * index_multiplier]*sin_u;
        a_imag =
            imag[(n + u*N/pow_2_i) % N * index_multiplier] +
            imag[(n + u*N/pow_2_i + N/pow_2_i) % N * index_multiplier]*cos_u +
            real[(n + u*N/pow_2_i + N/pow_2_i) % N * index_multiplier]*sin_u;

        g.sync();

        real[n * index_multiplier] = a_real;
        imag[n * index_multiplier] = a_imag;

        g.sync();
    }
}

void complex_fft(float *real, float *imag, const uint32_t N, const uint32_t index_multiplier) {
    dim3 threads_per_block(min(1024, N), 1);
    dim3 num_blocks(max(1, N/1024));
    void *params[4];
    params[0] = (void *)&real;
    params[1] = (void *)&imag;
    params[2] = (void *)&N;
    params[3] = (void *)&index_multiplier;
    hipLaunchCooperativeKernel(reinterpret_cast<const void*>(complex_fft_iteration), num_blocks, threads_per_block, params, 0, hipStreamDefault);
}

__global__ void untangle_and_pack(float *row1, float *row2, const uint32_t N,
                                  const uint32_t index_multiplier) {
    float Fr, Fi, Gr, Gi;
    grid_group g = this_grid();
    int i = g.thread_index().x+1;

    Fr = (row1[i * index_multiplier] + row1[(N-i) * index_multiplier]) / 2;
    Fi = (row2[i * index_multiplier] - row2[(N-i) * index_multiplier]) / 2;
    Gr = (row2[i * index_multiplier] + row2[(N-i) * index_multiplier]) / 2;
    Gi = -(row1[i * index_multiplier] - row1[(N-i) * index_multiplier]) / 2;

    row1[i * index_multiplier] = Fr;
    row1[(N-i) * index_multiplier] = Fi;
    row2[i * index_multiplier] = Gr;
    row2[(N-i) * index_multiplier] = Gi;
}

void real_pair_fft(float *row1, float *row2, const uint32_t len, const uint32_t index_multiplier) {
    complex_fft(row1, row2, len, index_multiplier);
    int N = len/2-1;
    dim3 threads_per_block(min(1024, N), 1);
    dim3 num_blocks(max(1, N/1024));
    untangle_and_pack<<<num_blocks, threads_per_block>>>(row1, row2, len, index_multiplier);
}

int matrix_fft(float *matrix, uint32_t rows, uint32_t cols) {
    float startTime = (float)clock()/CLOCKS_PER_SEC, endTime;
    if (!is_power_of_2(rows) || !is_power_of_2(cols)) {
        fprintf(stderr, "The image dimensions must be a power of 2.\n");
        return -1;
    }

    for (int row = 0; row < rows/2; row++) {
        real_pair_fft(matrix + row*cols, matrix + (row+rows/2)*cols, cols, 1);

    }

    real_pair_fft(matrix, matrix + cols/2, rows, cols);

    for (int col = 1; col < cols/2; col++) {
        complex_fft(matrix + col, matrix + cols-col, rows, cols);
    }

    endTime = (float)clock()/CLOCKS_PER_SEC;
    hipDeviceSynchronize();
    printf("%d by %d image FFT calculated in %f seconds.\n", rows, cols, endTime-startTime);
    return 0;
}

float *copy_matrix_to_gpu(float *host_matrix, int32_t rows, int32_t cols) {
    void *gpu_matrix;

    if (hipMalloc(&gpu_matrix, sizeof(*host_matrix)*rows*cols) != hipSuccess) {
        fprintf(stderr, "Unable to allocate memory for matrix on the GPU.");
        return NULL;
    }

    if (hipMemcpy(gpu_matrix, host_matrix, sizeof(*host_matrix)*rows*cols,
        hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Unable to copy image matrix on the GPU.");
        hipFree(gpu_matrix);
        return NULL;
    }

    return (float *)gpu_matrix;
}

int main() {
    uint32_t rows, cols;
    uint8_t *image_r, *image_g, *image_b;
    float *image_rf, *image_gf, *image_bf;

    if (read_shape(&rows, &cols) != 0) {
        fprintf(stderr, "Unable to read image shape.");
        return -1;
    }

    printf("Allocating memory.\n");
    image_r = (uint8_t *)malloc(rows*cols*sizeof(*image_r));
    if (image_r == NULL) {
        return -1;
    }
    image_g = (uint8_t *)malloc(rows*cols*sizeof(*image_g));
    if (image_g == NULL) {
        return -1;
    }
    image_b = (uint8_t *)malloc(rows*cols*sizeof(*image_b));
    if (image_b == NULL) {
        return -1;
    }

    if (get_color(image_r, rows, cols, IMG_R_FILENAME) != 0 ||
            get_color(image_g, rows, cols, IMG_G_FILENAME) != 0 ||
            get_color(image_b, rows, cols, IMG_B_FILENAME) != 0) {
        fprintf(stderr, "Unable to read image colors.");
        return -1;
    }

    image_rf = (float *)malloc(rows*cols*sizeof(*image_rf));
    if (image_rf == NULL) {
        return -1;
    }
    image_gf = (float *)malloc(rows*cols*sizeof(*image_gf));
    if (image_gf == NULL) {
        return -1;
    }
    image_bf = (float *)malloc(rows*cols*sizeof(*image_bf));
    if (image_bf == NULL) {
        return -1;
    }
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            image_rf[i*cols+j] = image_r[i*cols+j];
            image_gf[i*cols+j] = image_g[i*cols+j];
            image_bf[i*cols+j] = image_b[i*cols+j];
        }
    }

    float *gpu_image_rf, *gpu_image_gf, *gpu_image_bf;

    if ((gpu_image_rf = copy_matrix_to_gpu(image_rf, rows, cols)) == NULL) {
        return -1;
    }
    printf("GPU FFT:\n");
    if (matrix_fft(gpu_image_rf, rows, cols) != 0) {
        fprintf(stderr, "Error calculating fft.\n");
        return -1;
    }
    hipError_t cuda_error = hipMemcpy((void *)image_rf, (void *)gpu_image_rf, sizeof(*image_rf)*rows*cols,
                    hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "Unable to copy results from GPU to host. cuda error: %d\n", cuda_error);
        return -1;
    }
    printf("\n");
    hipFree(gpu_image_rf);

    if ((gpu_image_gf = copy_matrix_to_gpu(image_gf, rows, cols)) == NULL) {
        return -1;
    }
    if (matrix_fft(gpu_image_gf, rows, cols) != 0) {
        fprintf(stderr, "Error calculating fft.\n");
        return -1;
    }
    
    cuda_error = hipMemcpy((void *)image_gf, (void *)gpu_image_gf, sizeof(*image_gf)*rows*cols,
                    hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "Unable to copy results from GPU to host. cuda error: %d\n", cuda_error);
        return -1;
    }
    printf("\n");
    hipFree(gpu_image_gf);

    if ((gpu_image_bf = copy_matrix_to_gpu(image_bf, rows, cols)) == NULL) {
        return -1;
    }
    if (matrix_fft(gpu_image_bf, rows, cols) != 0) {
        fprintf(stderr, "Error calculating fft.\n");
        return -1;
    }
    cuda_error = hipMemcpy((void *)image_bf, (void *)gpu_image_bf, sizeof(*image_bf)*rows*cols,
                    hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "Unable to copy results from GPU to host. cuda error: %d\n", cuda_error);
        return -1;
    }
    printf("\n");
    hipFree(gpu_image_bf);

    printf("writing.\n");
    write_fft(image_rf, rows, cols, FFT_R_FILENAME);
    write_fft(image_gf, rows, cols, FFT_G_FILENAME);
    write_fft(image_bf, rows, cols, FFT_B_FILENAME);

    free(image_r);
    free(image_g);
    free(image_b);

    free(image_rf);
    free(image_gf);
    free(image_bf);

    return 0;
}